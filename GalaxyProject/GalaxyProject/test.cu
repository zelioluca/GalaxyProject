
#include <hip/hip_runtime.h>
/////*Author Luca Zelioli*/
////
////cuda inclusion 
//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <device_functions.h>
//#include <cuda_runtime_api.h>
//#include <device_atomic_functions.h>
////c++ iinclusion
//#include <iostream>
//#include <stdio.h>
//
////math library 
//#define _USE_MATH_DEFINES
//#include <math.h>
//#include <cmath>
//
//#include <string>
//#include <ctime> // maybe not needed 
//
////Histogram ORI 180
//#define HISTOGRAM_DEGREE 720
//
////define the thread in blocks
//#define THREADS_IN_BLOCKS 256
//
////project inclusion
//#include "function.h"; 
//
//using namespace std;
//
////Cuda error handling start here
//
//inline void error_check(cudaError_t err, const char*file, int line)
//{
//	if (err != cudaSuccess) {
//		::fprintf(stderr, "CUDA ERROR at %s[%d] : %s\n", file, line, cudaGetErrorString(err));
//		std::cout << "General error at %s[%d] : %s\n", file, line, cudaGetErrorString(err); 
//	}
//}
//
//#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)
//
//__global__ void DarkMatterMaster(int size, double* device_real, double* device_flat, long int* device_resultDD, long int* device_resultDR, long int* device_resultRR)
//{
//
//	//init all the histo to 0
//	for (int i = 0; i < HISTOGRAM_DEGREE; i++)
//	{
//		device_resultDD[i] = 0; 
//		device_resultDR[i] = 0; 
//		device_resultRR[i] = 0; 
//	}
//
//	//first thing copnvert in arc minutes to rad
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
// 
//	//qui inizia galaxy comp
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	int j = threadIdx.y + blockIdx.y * blockDim.y;
//	//int j = 0; 
//	if(i < size - 3)
//	{
//		//debug
//		// printf("GPU - j = %d, i =%d\n", j, i);
//		//allocate temp DD
//		double tempDD = acos(sin(device_real[j + 1]) * sin(device_real[i + 3]) + cos(device_real[j + 1])*cos(device_real[i + 3])*cos(device_real[j] - device_real[i + 2]));
//		//++device_resultDD[(int)(acos(sin(device_real[j + 1]) * sin(device_real[i + 3]) + cos(device_real[j + 1])*cos(device_real[i + 3])*cos(device_real[j] - device_real[i + 2])) * 180 / M_PI * 4)];
//		//debug
//		printf("TempDD -> %f\n", tempDD);
//		//allocate temp DR
//		double tempDR = acos(sin(device_real[j + 1]) * sin(device_flat[i + 1]) + cos(device_real[j + 1]) * cos(device_flat[i + 1]) * cos(device_real[j] - device_flat[i]));
//		//debug
//		printf("TempDR -> %f\n", tempDR);
//		//allocate temp RR
//		double tempRR = acos(sin(device_flat[j + 1]) * sin(device_flat[i + 3]) + cos(device_flat[j + 1])*cos(device_flat[i + 3])*cos(device_flat[j] - device_flat[i + 2]));
//		//debug
//		printf("TempRR -> %f\n", tempRR);
//		//atomicAdd
//		++device_resultDD[(int)(tempDD * 180 / M_PI * 4)];
//		++device_resultDR[(int)(tempDR * 180 / M_PI * 4)];
//		++device_resultRR[(int)(tempRR * 180 / M_PI * 4)];
//		//Test add 1 to go next couple of number CONTROLLARE
//		//i = i + 1; 
//		//j = j + 1; 
//		i += blockDim.x * gridDim.x;
//		j += blockDim.y * gridDim.y;
//
//	}
//
//	//sync the thread
//	__syncthreads();
//}
//
//int main(int argc, char* argv[])
//{
//	//start clock
//	clock_t masterStart = clock(); 
//	//device counter
//	int deviceCount = 0; 
//
//	//double host_real[5] = { 4646.98, 3749.51, 4644.35, 3749.52 };
//	//double host_flat[5] = { 840.961426, 387.991697, 387.368692, 2967.285746 };
//	double* host_real = CreateVector("Data/data_100k_arcmin.txt"); 
//	double* host_flat = CreateVector("Data/flat_100k_arcmin.txt");
//	size_t arraySize = GetVectorSize("Data/data_100k_arcmin.txt"); 
//	//size_t arraySize = 2; 
//
//	double* host_resultDD = (double*)malloc(HISTOGRAM_DEGREE * sizeof(double)); 
//	double* host_resultDR = (double*)malloc(HISTOGRAM_DEGREE * sizeof(double)); 
//	double* host_resultRR = (double*)malloc(HISTOGRAM_DEGREE * sizeof(double)); 
//
//	double * device_real; 
//	double * device_flat; 
//	long int * device_resultDD; 
//	long int * device_resultDR; 
//	long int * device_resultRR; 
//
//	if (cudaSuccess != cudaMalloc((void**)&device_real, arraySize * sizeof(double)))
//	{
//		std::cout << "Error in allocating device_real memory" << std::endl; 
//	}
//
//	if (cudaSuccess != cudaMalloc((void**)&device_flat, arraySize * sizeof(double)))
//	{
//		std::cout << "Error in allocating device_host memory" << std::endl;
//	}
//
//	if (cudaSuccess != cudaMalloc((void**)&device_resultDD, HISTOGRAM_DEGREE * sizeof(long int))) //usare int o lo ng int 
//	{
//		std::cout << "Error in allocating device_resultDD memory" << std::endl;
//	}
//
//	if (cudaSuccess != cudaMalloc((void**)&device_resultDR, HISTOGRAM_DEGREE * sizeof(long int)))
//	{
//		std::cout << "Error in allocating device_resultDR memory" << std::endl;
//	}
//
//	if (cudaSuccess != cudaMalloc((void**)&device_resultRR, HISTOGRAM_DEGREE * sizeof(long int)))
//	{
//		std::cout << "Error in allocating device_resultRR memory" << std::endl;
//	}
//
//	cudaMemcpy(device_real, host_real, arraySize * sizeof(double), cudaMemcpyHostToDevice); 
//	cudaMemcpy(device_flat, host_flat, arraySize * sizeof(double), cudaMemcpyHostToDevice);
//
//	//device props
//	//show device information
//	cudaGetDeviceCount(&deviceCount);
//	std::cout << "\nDevice count: " << deviceCount << std::endl;
//
//	//device property
//	for (int p = 0; p < deviceCount; p++)
//	{
//		cudaDeviceProp deviceProp;
//		cudaGetDeviceProperties(&deviceProp, p);
//
//		std::cout << "Device number " << p << std::endl;
//		std::cout << "\tDevice name " << deviceProp.name << std::endl;
//		std::cout << "\tMax threads dim " << deviceProp.maxThreadsDim << std::endl;;
//		std::cout << "\tMax grid size " << deviceProp.maxGridSize << std::endl;
//		std::cout << "\tMax threads per blocks " << deviceProp.maxThreadsPerBlock << std::endl;
//		std::cout << "\tShared memory per blocks " << deviceProp.sharedMemPerBlock << std::endl;
//		std::cout << "\tWarp size " << deviceProp.warpSize << std::endl;
//		std::cout << "\n"; 
//	}
//
//	//init set kernel
//	
//	//int blockInGrid = (arraySize + THREADS_IN_BLOCKS - 1) / THREADS_IN_BLOCKS;
//	//int limitX = 100000;
//	//int limitY = 100000;
//	//int threadLimitPerBlock = 140;
//	//int numberOfThreads = limitX * limitY;
//	//int requiredNumberOfBlocks = (numberOfThreads / threadLimitPerBlock) + 1;
// 
//	//Kernnel block and grid 
//	dim3 Grid(1, 1, 1); //348
//	dim3 ThreadsPerblock(1, 1, 1);  //288 block width 
//
//	//kernel 
//	DarkMatterMaster <<< Grid, ThreadsPerblock >>> (arraySize, device_real, device_flat, device_resultDD, device_resultDR, device_resultRR);
//	cudaDeviceSynchronize(); 
//	//error check
//	cudaError_t err = cudaGetLastError();
//	if (err != cudaSuccess)
//	{
//		std::cout << "Error in the kernel " << cudaGetErrorString(err) << std::endl;
//		return -1; 
//	}
//
//	//copy back the histogram array
//	cudaMemcpy(host_resultDD, device_resultDD, HISTOGRAM_DEGREE * sizeof(double), cudaMemcpyDeviceToHost);
//	cudaMemcpy(host_resultDR, device_resultDR, HISTOGRAM_DEGREE * sizeof(double), cudaMemcpyDeviceToHost);
//	cudaMemcpy(host_resultRR, device_resultRR, HISTOGRAM_DEGREE * sizeof(double), cudaMemcpyDeviceToHost);
//	//debug
//	
//	for (int i = 0; i < HISTOGRAM_DEGREE; i++)
//	{
//		std:cout << "hist DD=> " << i << " is "  << host_resultDD[i] << std::endl;  
//	}
//	//free c++ memory
//	std::cout << "Now frre the C++ memory\n" << std::endl; 
//	delete[] host_flat;
//	delete[] host_real; 
//	delete[] host_resultDD; 
//	delete[] host_resultDR; 
//	delete[] host_resultRR; 
//
//	//free cuda memory
//	std::cout << "Now frre the Cuda memory\n" << std::endl;
//	cudaFree(device_real); 
//	cudaFree(device_flat); 
//	cudaFree(device_resultDD); 
//	cudaFree(device_resultDR);
//	cudaFree(device_resultRR); 
//
//	//end clock
//	clock_t masterEnd = clock(); 
//	float elapsed = (float)(masterEnd - masterStart) / CLOCKS_PER_SEC; 
//	std::cout << "End procees in " << elapsed << " secs" << std::endl;
//
//
//	return EXIT_SUCCESS; 
//}