#include "hip/hip_runtime.h"
//cuda inclusion
#include "hip/hip_runtime.h"
#include ""
//c++ and project inclusion
#include <stdio.h>
#include "macro.h"

//Math inclusion
#define _USE_MATH_DEFINES
#include <math.h>
#include <cmath>

//Cuda error handling start here
inline void error_check(hipError_t err, const char*file, int line)
{
	if (err != hipSuccess) {
		::fprintf(stderr, "\nCUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(err));
		printf("\nGeneral error at %s[%d] : %s\n", file, line, hipGetErrorString(err));
	}
}

#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

//this function clamp the numbers ORI with float 
__device__ double ClampTheMatter(double temp, double a, double b)
{
	return fmax(a, fmin(b, temp));
}


__global__
void TheDarkMatter(double *device_real, double *device_flat, unsigned long long int *device_DD, unsigned long long int *device_DR, unsigned long long int *device_RR, int size, double *copy_device_real, double *copy_device_flat)
{

	//index for x and y era inverso c era x r era y
	const int cIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int rIndex = blockIdx.y * blockDim.y + threadIdx.y;

	int c = cIndex * 2; //ori c 
	int r = rIndex * 2; //ori r
 
	//check the bound of the two index 
	if ((c >= W ) || (r >= H )) return; 

	//this is the temp that contain DD
	double tempDD = sin(device_real[r + 1]) * sin(copy_device_real[c + 1]) + cos(device_real[r + 1]) * cos(copy_device_real[c + 1]) * cos(device_real[r] - copy_device_real[c]);
	//this is the temp for DR
	double tempDR = sin(device_real[r + 1]) * sin(device_flat[c + 1]) + cos(device_real[r + 1]) * cos(device_flat[c + 1]) * cos(device_real[r] - device_flat[c]);
	//this is the temp that contains RR
	double tempRR = sin(device_flat[r + 1]) * sin(copy_device_flat[c + 1]) + cos(device_flat[r + 1]) * cos(copy_device_flat[c + 1]) * cos(device_flat[r] - copy_device_flat[c]);
	//__syncthreads(); 
	//clamp the number DD
	tempDD = acos(ClampTheMatter(tempDD, MINCLAMP, MAXCLAMP));
	//this is the clamp for DR
	tempDR = acos(ClampTheMatter(tempDR, MINCLAMP, MAXCLAMP));
	//clamp the number RR
	tempRR = acos(ClampTheMatter(tempRR, MINCLAMP, MAXCLAMP));

	//calulate the stuff for the histo
	tempDD = (tempDD * 180 / M_PI * 4); 
	tempDR = (tempDR * 180 / M_PI * 4);
	tempRR = (tempRR * 180 / M_PI * 4); 
	//insert in the histogram DD ORI WITH (int)
	atomicAdd(device_DD + (int)tempDD, 1); 
	//this is the histogram DR
	atomicAdd(device_DR + (int)tempDR, 1);
	//insert in the histogram RR
	atomicAdd(device_RR + (int)tempRR, 1);

	__syncthreads();
}


void DarkMatterParallel(double *host_real, double *host_flat, unsigned long long int *host_DD, unsigned long long int *host_DR, unsigned long long int *host_RR, int size)
{
	//cuda malloc phase
	double *device_real = nullptr;
	double *device_flat = nullptr; 
	unsigned long long int *device_DD = nullptr;
	unsigned long long int *device_DR = nullptr;
	unsigned long long int *device_RR = nullptr; 

	//rookie level
	double *copy_device_real = nullptr;
	double *copy_device_flat = nullptr; 

	CUDA_CHECK(hipMalloc(&device_real, W * sizeof(double))); 
	CUDA_CHECK(hipMalloc(&device_flat, W * sizeof(double))); 
	CUDA_CHECK(hipMalloc(&device_DD, HISTOGRAM_DEGREE * sizeof(unsigned long long int)));
	CUDA_CHECK(hipMalloc(&device_DR, HISTOGRAM_DEGREE * sizeof(unsigned long long int)));
	CUDA_CHECK(hipMalloc(&device_RR, HISTOGRAM_DEGREE * sizeof(unsigned long long int)));

	//rookie 
	CUDA_CHECK(hipMalloc(&copy_device_real, W * sizeof(double)));
	CUDA_CHECK(hipMalloc(&copy_device_flat, W * sizeof(double)));

	//copy the array in the kernel
	CUDA_CHECK(hipMemcpy(device_real, host_real, W * sizeof(double), hipMemcpyHostToDevice)); 
	CUDA_CHECK(hipMemcpy(device_flat, host_flat, W * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_DD, host_DD, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_DR, host_DR, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_RR, host_RR, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyHostToDevice));


	//rookie 
	CUDA_CHECK(hipMemcpy(copy_device_real, host_real, W * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(copy_device_flat, host_flat, W * sizeof(double), hipMemcpyHostToDevice));

	//bulit the grid 
	const dim3 blockSize(TX, TY);
	const int bx = (NUMBERCASE + TX - 1) / TX; //W
	const int by = (NUMBERCASE + TY - 1) / TY; //50000 
	const dim3 gridSize = dim3(bx, by);
	
	printf("### Looking for DarkMatter ###\n\n"); 
	printf("\nGrid size is bx is %d by is %d \n\nNumber of case to analyze %ld x %ld\n\n TX is %d TY is %d\n\n", bx, by, NUMBERCASE, NUMBERCASE, TX, TY); 

	//start the kernel
	TheDarkMatter <<< gridSize, blockSize >>> (device_real, device_flat, device_DD, device_DR, device_RR, size, copy_device_real, copy_device_flat);
	hipDeviceSynchronize();
	//check error
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("\nError in cuda kernel %s\n", hipGetErrorString(err)); 

	}
	//copy result to host
	CUDA_CHECK(hipMemcpy(host_DD, device_DD, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(host_DR, device_DR, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(host_RR, device_RR, HISTOGRAM_DEGREE * sizeof(unsigned long long int), hipMemcpyDeviceToHost));

	//free the memory
	CUDA_CHECK(hipFree(device_real));
	CUDA_CHECK(hipFree(device_flat)); 
	CUDA_CHECK(hipFree(device_DD));
	CUDA_CHECK(hipFree(device_DR));
	CUDA_CHECK(hipFree(device_RR));

	//rookie
	CUDA_CHECK(hipFree(copy_device_real));
	CUDA_CHECK(hipFree(copy_device_flat));

}